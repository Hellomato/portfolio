
#include "hip/hip_runtime.h"

#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>;


const int hiddenNodes = 40;
const int cycles = 5000;
const float lr = 0.02f;
const int datapoints = 2000;
const float length = 0.4;




#define PI 3.14159265358979323846

struct point {
	float x;
	float y;
};

typedef struct {
	int width;
	int height;
	float* elements; // Matrices are stored in row - major order
} Matrix;

Matrix trainingElbows,trainingEndPoints,trainingAngles;
Matrix testingElbows, testingEndPoints, testingAngles;

float
randomf(const float min, const float max) {

	float num = rand() / (float)RAND_MAX;
	return min + num * (max - min);
}

//adaptation of code by Dr. Ian Howard
//howardlab.com
point* kinematics2D(float l1, float l2, float a1, float a2, float ox, float oy) {
	point points[2];

	point p1;
	p1.x = l1 * cos(a1) + 1.0f * ox;
	p1.y = l1 * sin(a1) + 1.0f * oy;

	point p2;
	p2.x = l1 * cos(a1) + l2 * cos(a1 + a2) + 1 * ox;
	p2.y = l1 * cos(a1) + l2 * cos(a1 + a2) + 1 * oy;

	points[0] = p1;
	points[1] = p2;

	return points;
}

void generatePoints() {
	time_t t;

	srand((unsigned)time(&t));
	//setup training matrices
	trainingAngles.height = 2;
	trainingAngles.width = datapoints;
	trainingAngles.elements = (float*)malloc(trainingAngles.width * trainingAngles.height * sizeof(float));
	
	trainingElbows.height = 2;
	trainingElbows.width = datapoints;
	trainingElbows.elements = (float*)malloc(trainingAngles.width * trainingAngles.height * sizeof(float));

	trainingEndPoints.height = 2;
	trainingEndPoints.width = datapoints;
	trainingEndPoints.elements = (float*)malloc(trainingAngles.width * trainingAngles.height * sizeof(float));

	//setup testing Matricies
	testingAngles.height = 2;
	testingAngles.width = datapoints;
	testingAngles.elements = (float*)malloc(trainingAngles.width * trainingAngles.height * sizeof(float));
	testingElbows.height = 2;
	testingElbows.width = datapoints;
	testingElbows.elements = (float*)malloc(trainingAngles.width * trainingAngles.height * sizeof(float));
	testingEndPoints.height = 2;
	testingEndPoints.width = datapoints;
	testingEndPoints.elements = (float*)malloc(trainingAngles.width * trainingAngles.height * sizeof(float));

	for (int i = 0; i < datapoints; i++) {
		

		float a1 = randomf(0.0, PI);
		float a2 = randomf(0.0, PI);
		
		trainingAngles.elements[i] = a1;
		trainingAngles.elements[i + trainingAngles.width] = a2;

		float a3 = randomf(0.0, PI);
		float a4 = randomf(0.0, PI);

		testingAngles.elements[i] = a3;
		testingAngles.elements[i + testingAngles.width] = a4;

		point* data = kinematics2D(length, length, a1, a2, 0, 0);

		trainingEndPoints.elements[i] = data[0].x;
		trainingEndPoints.elements[i + trainingEndPoints.width] = data[0].y;
		
		trainingElbows.elements[i] = data[1].x;
		trainingElbows.elements[i + trainingElbows.width] = data[1].y;
		
		data = kinematics2D(length, length, a3, a4, 0, 0);

		testingEndPoints.elements[i] = data[0].x;
		testingEndPoints.elements[i + testingEndPoints.width] = data[0].y;

		testingEndPoints.elements[i] = data[1].x;
		testingEndPoints.elements[i + testingEndPoints.width] = data[1].y;
	}
	
}

//adapted from practical 6
Matrix MatrixMultiplicationLinear(Matrix A,Matrix B) {
	Matrix C;
	C.height = A.height;
	C.width = B.width;
	C.elements = (float*)malloc(C.width * C.height * sizeof(float));
	for (int i = 0; i < C.height; i++) {
		for (int j = 0; j < C.width; j++) {
			C.elements[j + i * C.width] = 0;

			for (int k = 0; k < A.width; k++) {
				C.elements[j + i * C.width] += A.elements[k + i * A.width] * B.elements[j + k * B.width];
			}
		}

	}

	return C;
}

Matrix transpose(Matrix A) {
	Matrix B;
	B.height = A.width;
	B.width = A.height;
	B.elements = (float*)malloc(B.width * B.height * sizeof(float));
	for (int i = 0; i < B.width; ++i)
		for (int j = 0; j < B.height; ++j) {
			B.elements[i + j * B.width] = A.elements[j + i * A.width];
		}
	return B;
}
float Average(Matrix A) {
	float sum = 0;
	float average;
	for (int i = 0; i < A.height; i++) {
		for (int j = 0; j < A.width; j++) {
			sum += A.elements[j + i * A.width];
		}
	}
	average = sum / (float)(A.height * A.width);
	return average;
}

void printMatrix(Matrix A) {
	for (int k = 0; k < A.height; k++) {
		for (int l = 0; l < A.width; l++) {
			printf("%f ", A.elements[k * A.width + l]);
		}
		printf(";\n");
	}
}
void linearSolution() {

	//number of inputs and outputs for network
	const int inputs = 3;
	const int outputs = 2;

	//setup network arrays
	Matrix w1;
	w1.height = hiddenNodes;
	w1.width = inputs;
	w1.elements = (float*)malloc(w1.width * w1.height * sizeof(float));
	Matrix w2;
	w2.height = outputs;
	w2.width = hiddenNodes + 1;
	w2.elements = (float*)malloc(w2.width * w2.height * sizeof(float));
	
	//initalise weight matrix
	for (int i = 0; i < w1.height; i++) {
		for (int j = 0; j < w1.width; j++) {
			w1.elements[j + i * w1.width]  = randomf(0.f, 1.0f);

		}

	}
	for (int k = 0; k < w2.height; k++) {
		for (int i = 0; i < w2.width; i++) {
			w2.elements[i + k * w2.width] = randomf(0.f, 1.0f);
		}
		
	}



	//train network
	//for n cycles
	Matrix error{1,cycles};
	error.elements = (float*)malloc(error.width * error.height * sizeof(float));
	for (int i = 0; i < cycles; i++) {
		//for each datapoint
		for (int j = 0; j < datapoints; j++) {
			
			//initialise variables for feed Forward
			Matrix l1activation;
			Matrix activation;

			//get current data and augment
			Matrix currentData = { 1,inputs };
			currentData.elements = (float*)malloc(currentData.width * currentData.height * sizeof(float));
			currentData.elements[0] = trainingEndPoints.elements[i];
			currentData.elements[1] = trainingEndPoints.elements[i + trainingEndPoints.width];
			currentData.elements[2] = 1;
			//activations for layer 1
			l1activation = MatrixMultiplicationLinear(w1, currentData);

			//sigmoid and augment agian
			Matrix l1Augment = { l1activation.width, l1activation.height + 1};
			
			l1Augment.elements = (float*)malloc(l1Augment.width * l1Augment.height * sizeof(float));
			
			for (int k = 0; k < l1Augment.height * l1Augment.width; k++) {
				if (k < l1activation.height * l1activation.width) {
					
					l1Augment.elements[k] = 1.0f / (float) (1 + exp(-l1activation.elements[k]));
				}
				else {
					l1Augment.elements[k] = 1;
				}
				
			}

			//get final activations
			activation = MatrixMultiplicationLinear(w2, l1Augment);
			///////////////////////////////////////////////////////////
			
			//calculate delta values
			Matrix d2 = {1,hiddenNodes};
			d2.elements = (float*)malloc(d2.width * d2.height * sizeof(float));
			Matrix d3 = { 1,outputs };
			d3.elements = (float*)malloc(d3.width * d3.height * sizeof(float));
			//get current targets
			float t[2] = { trainingAngles.elements[j],trainingAngles.elements[j + trainingAngles.width] };
			
			for (int k = 0; k < outputs; k++) {
				d3.elements[k] = -(t[k] - activation.elements[k]);
				
			}
			Matrix w2t = transpose(w2);
			for (int k = 0; k < hiddenNodes; k++) {
				Matrix temp = MatrixMultiplicationLinear(w2t, d3);
				d2.elements[k] = temp.elements[k] * (l1Augment.elements[k] * (1 - l1Augment.elements[k]));
				free(temp.elements);
			}
			//////////////////////////////////////////////////////////////
			//update weights

			//update w1
			for (int k = 0; k < w1.height; k++) {
				for (int l = 0; l < w1.width; l++) {
					Matrix inputDataT = transpose(currentData);
					
					Matrix d2xInputT = MatrixMultiplicationLinear(d2, inputDataT);
					w1.elements[l + k * w1.width] = w1.elements[l + k * w1.width] - lr * d2xInputT.elements[l + k * d2xInputT.width];
					free(inputDataT.elements);
					free(d2xInputT.elements);
				}
			}
			//update w2
			for (int k = 0; k < w2.height; k++) {
				for (int l = 0; l < w2.width; l++) {
					Matrix a2T = transpose(l1Augment);
					Matrix d3xa2t = MatrixMultiplicationLinear(d3, a2T);
					w2.elements[l + k * w2.width] = w2.elements[l + k * w2.width] - lr * d3xa2t.elements[l + k * w2.width];
					free(a2T.elements);
					free(d3xa2t.elements);
				}
			}
			free(currentData.elements);
			free(activation.elements);
			free(d2.elements);
			free(d3.elements);
			free(w2t.elements);
			free(l1Augment.elements);
			free(l1activation.elements);
			
		}


		Matrix activationCycle;
			//initialise variables for feed Forward
			Matrix l1activation;

			//get current data and augment
			Matrix currentData = trainingEndPoints;
			
			//activations for layer 1
			l1activation = MatrixMultiplicationLinear(w1, currentData);


			//sigmoid and augment agian
			Matrix l1Augment = { l1activation.width, l1activation.height + 1};
			
			l1Augment.elements = (float*)malloc(l1Augment.width * l1Augment.height * sizeof(float));
			
			for (int k = 0; k < l1Augment.height * l1Augment.width; k++) {
				if (k < l1activation.height * l1activation.width) {
					
					l1Augment.elements[k] = 1.0f / (1 + exp(-l1activation.elements[k]));
				}
				else {
					l1Augment.elements[k] = 1;
				}
				
			}
			//get final activations
			activationCycle = MatrixMultiplicationLinear(w2, l1Augment);
			
			for (int k = 0; k < activationCycle.width * activationCycle.height; k++) {
				activationCycle.elements[k] = (trainingAngles.elements[k] - activationCycle.elements[k]) * (trainingAngles.elements[k] - activationCycle.elements[k]);
			}

		float average = Average(activationCycle);
		error.elements[i] = average;
		free(activationCycle.elements);
		free(l1Augment.elements);
		free(l1activation.elements);
		printf("e(%d): %f \n",i, error.elements[i]);
	}

}


int main()
{
	generatePoints();

	linearSolution();
	


}
