#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <time.h>;
#include <stdlib.h>
#include <stdio.h>
#include"win-gettimeofday.h" //taken from practical work, all yours Dr Gianni


#define BLOCK_SIZE 32
#define DIMX 1000
#define DIMY 1000
#define DIMZ 1000

typedef struct {
	int width;
	int height;
	float* elements; 
} Matrix;



Matrix MatrixMultiplicationLinear(Matrix A, Matrix B) {
	Matrix C;
	C.height = A.height;
	C.width = B.width;
	//allocate memory
	C.elements = (float*)malloc(C.width * C.height * sizeof(float));
	//for each point in matrix C 
	for (int i = 0; i < C.height; i++) {
		for (int j = 0; j < C.width; j++) {
			C.elements[j + i * C.width] = 0;
			
			//calculate value
			for (int k = 0; k < A.width; k++) {
				C.elements[j + i * C.width] += A.elements[k + i * A.width] * B.elements[j + k * B.width];
			}
		}

	}

	return C;
}

__global__ void matrixMult(Matrix A, Matrix B, Matrix C) {

	//variables for shared memory
	__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	//block ids
	int blockY = blockIdx.y;
	int blockX = blockIdx.x;

	float sum = 0.f;
	//thread ids
	int threadY = threadIdx.y;
	int threadX = threadIdx.x;

	//fill As and Bs
	for (int i = 0; i < (BLOCK_SIZE + A.width - 1) / BLOCK_SIZE; i++) {
		//if thread should have a value (if matrix dimentions dont match blocksize)
		if (i * BLOCK_SIZE + threadX < A.width && blockY * BLOCK_SIZE + threadY < A.height) {
			As[threadY][threadX] = A.elements[(blockY * BLOCK_SIZE + threadY) * A.width + i * BLOCK_SIZE + threadX];
		}
		else
		{
			As[threadY][threadX] = 0.f;
		}
		//same for Bs
		if (i * BLOCK_SIZE + threadY < B.height && blockX * BLOCK_SIZE + threadX < B.width) {
			Bs[threadY][threadX] = B.elements[(i * BLOCK_SIZE + threadY) * B.width + blockX * BLOCK_SIZE + threadX];
		}
		else
		{
			Bs[threadY][threadX] = 0.f;
		}
		//make sure everything is done
		__syncthreads();

		//work out partial sums
		for (int j = 0; j < BLOCK_SIZE; j++)
			sum += As[threadY][j] * Bs[j][threadX];

		__syncthreads();
	}
	//check if threadX/y is a position in C.elements (same as before)
	if (blockY * BLOCK_SIZE + threadY < C.height && blockX * BLOCK_SIZE + threadX < C.width) {
		
		C.elements[((blockY * BLOCK_SIZE + threadY) * C.width) + (blockX * BLOCK_SIZE) + threadX] = sum;
	}

}

Matrix cudaSolution(const Matrix A, const Matrix B) {

	//create and allocate memeory for devices
	Matrix DeviceA;
	Matrix DeviceB;
	Matrix DeviceC;
	Matrix C;
	DeviceC.width = C.width = B.width;
	DeviceC.height = C.height = A.height;

	C.elements = (float*)malloc(C.width * C.height * sizeof(float));

	DeviceA.height = A.height;
	DeviceA.width = A.width;

	hipMalloc(&DeviceA.elements, A.width * A.height * sizeof(float));
	//copy A to deviceA
	hipMemcpy(DeviceA.elements, A.elements, A.width * A.height * sizeof(float), hipMemcpyHostToDevice);

	DeviceB.width = B.width;
	DeviceB.height = B.height;
	hipMalloc(&DeviceB.elements, A.width * A.height * sizeof(float));
	//copy B to deviceB
	hipMemcpy(DeviceB.elements, B.elements, B.width * B.height * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&DeviceC.elements, C.width * C.height * sizeof(float));
	
	
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//set gridsize
	dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x, (A.height + dimBlock.y - 1) / dimBlock.y);

	matrixMult << <dimGrid, dimBlock >> > (DeviceA, DeviceB, DeviceC);
	//make sure everything finished
	hipDeviceSynchronize();

	///copy finished matrix to output matrix
	hipMemcpy(C.elements, DeviceC.elements, C.width * C.height * sizeof(float), hipMemcpyDeviceToHost);
	
	
	hipFree(DeviceA.elements);
	hipFree(DeviceB.elements);
	hipFree(DeviceC.elements);
	
	return C;
}

void initialiseMatrices(Matrix &A, Matrix &B) {
	//stuff for random
	time_t t;

	srand((unsigned)time(&t));
	A.elements = (float*)malloc(A.width * A.height * sizeof(float));
	B.elements = (float*)malloc(B.width * B.height * sizeof(float));

	//fill both matrices with random numbers 0-10
	for (int i = 0; i < A.height; i++) {
		for (int j = 0; j < A.width; j++) {
			A.elements[j + i * A.width] = rand() % 10;
		}
	}
	for (int i = 0; i < B.height; i++) {
		for (int j = 0; j < B.width; j++) {
			B.elements[j + i * B.width] = rand() % 10;
		}
	}

}

int main(int argc,char*argv[])
{
	Matrix A;
	Matrix B;
	//get matrix dimensions
	printf("Enter height for Matrix 1 \n");
	scanf("%d",&A.height);
	printf("Enter width for Matrix 1 \n");
	scanf("%d", &A.width);

	printf("Enter height for Matrix 2 \n");
	scanf("%d", &B.height);
	printf("Enter width for Matrix 2 \n");
	scanf("%d", &B.width);

	while (A.width != B.height) {
		printf("invalid dimentions \n");
		printf("Enter height for Matrix 1 \n");
		scanf("%d", &A.height);
		printf("Enter width for Matrix 1 \n");
		scanf("%d", &A.width);

		printf("Enter height for Matrix 2 \n");
		scanf("%d", &B.height);
		printf("Enter width for Matrix 2 \n");
		scanf("%d", &B.width);
	}

	initialiseMatrices(A,B);
	//get time
	double start = 0.0;
	start = get_current_time();
	//do linear multiplication
	Matrix C = MatrixMultiplicationLinear(A, B);
	//get time again
	double  end = 0.0;
	end = get_current_time();
	//work out how much time passed
	double t = (end - start);
	
	printf("Time Taken for linear:  %lf seconds \n", t);

	start = get_current_time();
	//do paralel multiplication
	Matrix PC = cudaSolution(A,B);
	
	end = get_current_time();

	t = (end - start);
	printf("Time Taken for paralel:  %lf seconds \n", t);


	
	bool isSame = true;
	for (int i = 0; i < PC.height; i++) {
		for (int j = 0; j < PC.width; j++) {

			
			if (PC.elements[i * PC.width + j] != C.elements[i * PC.width + j]) {
				isSame = false;
				printf("%d, %f \n", i * PC.width + j,PC.elements[i * PC.width + j] - C.elements[i * PC.width + j]);
			}

		}
	}

	if (isSame) {
		printf("Both Matrices are the same");
	}
	else {
		printf("Matrices do not match");
	}

	free(A.elements);
	free(B.elements);
	free(C.elements);
	free(PC.elements);
    return 0;
}


